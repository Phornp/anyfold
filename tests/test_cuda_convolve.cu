#include "hip/hip_runtime.h"
#define BOOST_TEST_DYN_LINK 
#define BOOST_TEST_MODULE CUDA_CONVOLUTION
#include "boost/test/unit_test.hpp"
#include "test_fixtures.hpp"
#include <numeric>
#include <vector>
#include <functional>
//#include "padd_utils.h"
#include "cuda_convolve.cuh"
#include "cuda_helpers.cuh"


BOOST_AUTO_TEST_CASE( trivial_convolve )
{

  using namespace cuda_convolve;

  static const unsigned kernel_axis_length = 3;
  static const unsigned image_axis_length = 64+2*(kernel_axis_length); 
  static const unsigned num_pixels = image_axis_length*image_axis_length*image_axis_length;
  static const unsigned num_k_pixels = kernel_axis_length*kernel_axis_length*kernel_axis_length;

  std::vector<float> output(num_pixels);
  std::vector<float> input (num_pixels);
  std::vector<float> kernel(num_k_pixels);

  std::fill(kernel.begin(), kernel.end(),0.f);
  std::fill(input.begin(),input.end(),10); 
  
  unsigned long checksum = num_pixels*10;

  float* d_input = 0;
  float* d_kernel = 0;
  float* d_output = 0;
  
  //allocat ememory in GPU
  HANDLE_ERROR( hipMalloc( (void**)&(d_input), num_pixels*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&(d_output), num_pixels*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&(d_kernel), (num_k_pixels)*sizeof(float) ) );

  HANDLE_ERROR( hipMemcpy( d_input, &input[0] , num_pixels*sizeof(float) , hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( d_output, &output[0] , num_pixels*sizeof(float) , hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( d_kernel, &kernel[0] , num_k_pixels*sizeof(float) , hipMemcpyHostToDevice ) );

  dim3 threads(128);
  dim3 blocks((num_pixels + threads.x -1)/threads.x);

  uint3 padded_image_dims;
  padded_image_dims.x  = image_axis_length;
  padded_image_dims.y  = image_axis_length;
  padded_image_dims.z  = image_axis_length;

  static_convolve<3u><<<threads,blocks>>>(d_input, d_kernel, d_output, padded_image_dims);

  HANDLE_ERROR( hipMemcpy( &output[0], d_output , output.size()*sizeof(float) , hipMemcpyDeviceToHost ) );

  unsigned long sum = std::accumulate(output.begin(),output.end(),0.f);
  BOOST_CHECK_LT(sum, checksum);

  HANDLE_ERROR( hipFree( d_input ) );
  HANDLE_ERROR( hipFree( d_kernel ) );
  HANDLE_ERROR( hipFree( d_output ) );

}

// BOOST_AUTO_TEST_CASE( identity_convolve )
// {
  
//   using namespace cuda_convolve;

//   float sum_expected = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);

//   zero_padd<image_stack> padder(image_.shape(), identity_kernel_.shape());
//   image_stack padded_image(padder.extents_, image_.storage_order());
//   padder.insert_at_offsets(image_, padded_image);
  
//   std::vector<int> extents_as_int(padder.extents_.size());
//   std::copy(padder.extents_.begin(), padder.extents_.end(), extents_as_int.begin());

//   convolution3DfftCUDAInPlace(padded_image.data(), &extents_as_int[0], 
//   			  identity_kernel_.data(),&kernel_dims_[0],
//   			  selectDeviceWithHighestComputeCapability());

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   BOOST_CHECK_CLOSE(sum, sum_expected, .00001);


// }

// BOOST_AUTO_TEST_CASE( horizontal_convolve )
// {
//   using namespace cuda_convolve;

//   float sum_expected = std::accumulate(image_folded_by_horizontal_.data(), image_folded_by_horizontal_.data() + image_folded_by_horizontal_.num_elements(),0.f);

//   zero_padd<image_stack> padder(image_.shape(), horizont_kernel_.shape());
//   image_stack padded_image(padder.extents_, image_.storage_order());

//   padder.insert_at_offsets(image_, padded_image);
  
//   std::vector<int> extents_as_int(padder.extents_.size());
//   std::copy(padder.extents_.begin(), padder.extents_.end(), extents_as_int.begin());

//   convolution3DfftCUDAInPlace(padded_image.data(), &extents_as_int[0], 
// 			      horizont_kernel_.data(),&kernel_dims_[0],
// 			      selectDeviceWithHighestComputeCapability());

//   image_ = padded_image[ boost::indices[range(padder.offsets()[0], padder.offsets()[0]+image_dims_[0])][range(padder.offsets()[1], padder.offsets()[1]+image_dims_[1])][range(padder.offsets()[2], padder.offsets()[2]+image_dims_[2])] ];
  
//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);

//     BOOST_REQUIRE_CLOSE(sum, sum_expected, .00001);
 
// }

// BOOST_AUTO_TEST_CASE( vertical_convolve )
// {
  
//   multiviewnative::zero_padd<multiviewnative::image_stack> padder(image_.shape(), vertical_kernel_.shape());
//   multiviewnative::image_stack padded_image(padder.extents_, image_.storage_order());

//   padder.insert_at_offsets(image_, padded_image);
  
//   std::vector<int> extents_as_int(padder.extents_.size());
//   std::copy(padder.extents_.begin(), padder.extents_.end(), extents_as_int.begin());

//   convolution3DfftCUDAInPlace(padded_image.data(), &extents_as_int[0], 
// 			      vertical_kernel_.data(),&kernel_dims_[0],
// 			      selectDeviceWithHighestComputeCapability());


//   float sum_expected = std::accumulate(image_folded_by_vertical_.data(), image_folded_by_vertical_.data() + image_folded_by_vertical_.num_elements(),0.f);

//   image_ = padded_image[ boost::indices[multiviewnative::range(padder.offsets()[0], padder.offsets()[0]+image_dims_[0])][multiviewnative::range(padder.offsets()[1], padder.offsets()[1]+image_dims_[1])][multiviewnative::range(padder.offsets()[2], padder.offsets()[2]+image_dims_[2])] ];

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   BOOST_CHECK_CLOSE(sum, sum_expected, .00001);


// }

// BOOST_AUTO_TEST_CASE( depth_convolve )
// {
  
//   multiviewnative::zero_padd<multiviewnative::image_stack> padder(image_.shape(), depth_kernel_.shape());
//   multiviewnative::image_stack padded_image(padder.extents_, image_.storage_order());

//   padder.insert_at_offsets(image_, padded_image);
  
//   std::vector<int> extents_as_int(padder.extents_.size());
//   std::copy(padder.extents_.begin(), padder.extents_.end(), extents_as_int.begin());

//   convolution3DfftCUDAInPlace(padded_image.data(), &extents_as_int[0], 
// 			      depth_kernel_.data(),&kernel_dims_[0],
// 			      selectDeviceWithHighestComputeCapability());


//   float sum_expected = std::accumulate(image_folded_by_depth_.data(), image_folded_by_depth_.data() + image_folded_by_depth_.num_elements(),0.f);

//   image_ = padded_image[ boost::indices[multiviewnative::range(padder.offsets()[0], padder.offsets()[0]+image_dims_[0])][multiviewnative::range(padder.offsets()[1], padder.offsets()[1]+image_dims_[1])][multiviewnative::range(padder.offsets()[2], padder.offsets()[2]+image_dims_[2])] ];

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   BOOST_CHECK_CLOSE(sum, sum_expected, .00001);


// }

// BOOST_AUTO_TEST_CASE( all1_convolve )
// {
  
//   multiviewnative::zero_padd<multiviewnative::image_stack> padder(image_.shape(), all1_kernel_.shape());
//   multiviewnative::image_stack padded_image(padder.extents_, image_.storage_order());

//   padder.insert_at_offsets(image_, padded_image);
  
//   std::vector<int> extents_as_int(padder.extents_.size());
//   std::copy(padder.extents_.begin(), padder.extents_.end(), extents_as_int.begin());

//   convolution3DfftCUDAInPlace(padded_image.data(), &extents_as_int[0], 
// 			      all1_kernel_.data(),&kernel_dims_[0],
// 			      selectDeviceWithHighestComputeCapability());


//   float sum_expected = std::accumulate(image_folded_by_all1_.data(), image_folded_by_all1_.data() + image_folded_by_all1_.num_elements(),0.f);

//   image_ = padded_image[ boost::indices[multiviewnative::range(padder.offsets()[0], padder.offsets()[0]+image_dims_[0])][multiviewnative::range(padder.offsets()[1], padder.offsets()[1]+image_dims_[1])][multiviewnative::range(padder.offsets()[2], padder.offsets()[2]+image_dims_[2])] ];

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   BOOST_CHECK_CLOSE(sum, sum_expected, .00001);


// }

// BOOST_AUTO_TEST_SUITE_END()

// BOOST_FIXTURE_TEST_SUITE( gpu_convolution_works, multiviewnative::default_3D_fixture )

// BOOST_AUTO_TEST_CASE( trivial_convolve_newapi )
// {
//   using namespace cuda_convolve;
  
//   float* kernel = new float[kernel_size_];
//   std::fill(kernel, kernel+kernel_size_,0.f);

//   image_stack expected = image_;
//   std::fill(expected.data(), expected.data() + expected.num_elements(),0.f);

//   inplace_gpu_convolution(image_.data(), &image_dims_[0], 
// 			  kernel,&kernel_dims_[0],
// 			  selectDeviceWithHighestComputeCapability());


//   float sum = std::accumulate(image_.data(), image_.data() + image_size_,0.f);
//   try{
//     BOOST_REQUIRE_CLOSE(sum, 0.f, .00001);
//   }
//   catch(...){
//     std::cout << "expected:\n" << expected << "\n"
// 	      << "received:\n" << image_ << "\n";
//   }

//   delete [] kernel;
// }

// BOOST_AUTO_TEST_CASE( identity_convolve_newapi )
// {
//   using namespace cuda_convolve;

  
//   image_stack expected = image_;
//   float sum_original = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   inplace_gpu_convolution(image_.data(), &image_dims_[0], 
//   			  identity_kernel_.data(),&kernel_dims_[0],
//   			  selectDeviceWithHighestComputeCapability());

  

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   try{
//     BOOST_REQUIRE_CLOSE(sum, sum_original, .00001);
//   }
//   catch(...){
//     std::cout << boost::unit_test::framework::current_test_case().p_name << "\n"
// 	      << "expected:\n" << expected << "\n"
// 	      << "received:\n" << image_ << "\n";
//   }


// }

// BOOST_AUTO_TEST_CASE( horizontal_convolve_newapi )
// {
//   using namespace cuda_convolve;

  

//   float sum_original = std::accumulate(image_folded_by_horizontal_.data(), image_folded_by_horizontal_.data() + image_.num_elements(),0.f);
//   inplace_gpu_convolution(image_.data(), &image_dims_[0], 
//   			  horizont_kernel_.data(),&kernel_dims_[0],
//   			  selectDeviceWithHighestComputeCapability());

  

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   //BOOST_CHECK_CLOSE(sum, sum_original, .00001);
//   try{
//     BOOST_REQUIRE_CLOSE(sum, sum_original, .00001);
//   }
//   catch(...){
//     std::cout << boost::unit_test::framework::current_test_case().p_name << "\n"
// 	      << "expected:\n" << image_folded_by_horizontal_ << "\n"
// 	      << "received:\n" << image_ << "\n";
//   }

// }

// BOOST_AUTO_TEST_CASE( vertical_convolve_newapi )
// {
//   using namespace cuda_convolve;

  

//   float sum_original = std::accumulate(image_folded_by_vertical_.data(), image_folded_by_vertical_.data() + image_.num_elements(),0.f);
//   inplace_gpu_convolution(image_.data(), &image_dims_[0], 
//   			  vertical_kernel_.data(),&kernel_dims_[0],
//   			  selectDeviceWithHighestComputeCapability());

  

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   // BOOST_CHECK_CLOSE(sum, sum_original, .00001);
//  try{
//     BOOST_REQUIRE_CLOSE(sum, sum_original, .00001);
//   }
//   catch(...){
//     std::cout << boost::unit_test::framework::current_test_case().p_name << "\n" 
// 	      << "expected:\n" << image_folded_by_vertical_ << "\n"
// 	      << "received:\n" << image_ << "\n";
//   }

// }

// BOOST_AUTO_TEST_CASE( all1_convolve_newapi )
// {
//   using namespace cuda_convolve;

  

//   float sum_original = std::accumulate(image_folded_by_all1_.data(), image_folded_by_all1_.data() + image_.num_elements(),0.f);
//   inplace_gpu_convolution(image_.data(), &image_dims_[0], 
//   			  all1_kernel_.data(),&kernel_dims_[0],
//   			  selectDeviceWithHighestComputeCapability());

  

//   float sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(),0.f);
//   // BOOST_CHECK_CLOSE(sum, sum_original, .00001);
// try{
//     BOOST_REQUIRE_CLOSE(sum, sum_original, .00001);
//   }
//   catch(...){
//     std::cout << boost::unit_test::framework::current_test_case().p_name << "\n"
// 	      << "expected:\n" << image_folded_by_all1_ << "\n"
// 	      << "received:\n" << image_ << "\n";
//   }


// }
//BOOST_AUTO_TEST_SUITE_END()
